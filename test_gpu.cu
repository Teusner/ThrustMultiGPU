#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>


int main(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;

    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
    }

    if (deviceCount > 2) {
        unsigned int size = 1000;
        hipSetDevice(0);
        thrust::device_vector<float> a(size, 2);
        thrust::device_vector<float> c(size);

        hipSetDevice(1);
        thrust::device_vector<float> b(size, 1);
        thrust::device_vector<float> d(size);

        hipSetDevice(0);
        thrust::transform(a.begin(), a.end(), b.begin(), c.begin(), thrust::plus<float>());

        hipSetDevice(1);
        thrust::transform(a.begin(), a.end(), b.begin(), d.begin(), thrust::minus<float>());
    }
}